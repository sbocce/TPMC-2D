#include "hip/hip_runtime.h"
// Copyright (c) 2020, Stefano Boccelli
// All rights reserved.
// 
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
// * Redistributions of source code must retain the above copyright notice, 
//   this list of conditions and the following disclaimer.
// * Redistributions in binary form must reproduce the above copyright notice, 
//   this list of conditions and the following disclaimer in the documentation 
//   and/or other materials provided with the distribution.
// * Neither the name of the copyright holder's organization nor the names of 
//   its contributors may be used to endorse or promote products derived from 
//   this software without specific prior written permission.
// 
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" 
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE 
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE 
// ARE DISCLAIMED. IN NO EVENT SHALL S. BOCCELLI BE LIABLE FOR ANY DIRECT, 
// INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES 
// (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; 
// LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND 
// ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT 
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF 
// THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//
////////////////////////////////////////////////////////////////////////////////
//
// You can find a description of the algorithm in:
// "Numerical Investigation of Reversed Gas Feed Configurations for Hall 
//  Thrusters", S. Boccelli, T.E. Magin, A. Frezzotti (submitted, 2020).
//
////////////////////////////////////////////////////////////////////////////////

#include <iostream>
#include <fstream>
#include <math.h>
#include <time.h>
#include <string>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define PI 3.1415926535
#define KB 1.38064852E-23

// typedef double fpTYPE; // Floating point type. Defined here for fast switching from double to float in case of GPU.
typedef float fpTYPE; // Floating point type. Defined here for fast switching from double to float in case of GPU.

#include "ReadFiles_cuda.h"


// ############################################################################

__global__ void initRNG_kernel(hiprandState *state)
{
  int th_id = threadIdx.x + blockIdx.x*blockDim.x;
  hiprand_init(12345, th_id, 0, &state[th_id]);
}

// ############################################################################

__device__ double rnum(hiprandState *state)
{
  int th_id = threadIdx.x + blockIdx.x*blockDim.x;

  // return ((double) rand() / (RAND_MAX));
  return hiprand_uniform_double(&state[th_id]);
}

// ############################################################################

__device__ void maxwellian(fpTYPE UX, fpTYPE UY, fpTYPE UZ, fpTYPE T, 
                           fpTYPE M, hiprandState *state, fpTYPE* vel_new)
{
  // Computes three velocities from a drifted Maxwellian

  for(size_t i = 0; i < 3; ++i)
  {
    // Step 1.
    fpTYPE THETA = 2.0*PI*rnum(state);

    // Step 2.
    fpTYPE BETA = 1.0/sqrt(2.0*KB/M*T);

    // R may go from 0 to 1 included. Remove the extremes in a very rough way
    // or the log() will explode
    vel_new[i] = sin(THETA)*sqrt(-log(rnum(state)*0.9999999 + 1.0e-10))/BETA;
  }

  // Step 3. Add average velocity
  vel_new[0] += UX;
  vel_new[1] += UY;
  vel_new[2] += UZ;

  return;

}

// #####################################################

__device__ fpTYPE psi_HS(fpTYPE x)
{
// This is an auxiliary function for the computation of HS collisions 
// (see Cercignani, The Boltzmann equation and its applications (1988), pag. 179).

  return exp(-x*x) + (2.0*x + 1.0/(x + 0.000000001))*sqrt(PI)/2.0*erf(x);
}

// #####################################################

__device__ void interp2D(fpTYPE xP, fpTYPE yP, 
                         fpTYPE* xx, 
                         fpTYPE* yy,
                         fpTYPE* N_MAT, 
                         fpTYPE* V1_MAT, 
                         fpTYPE* V2_MAT, 
                         fpTYPE* V3_MAT, 
                         fpTYPE* T_MAT,
                         size_t Nx, size_t Ny,
                         fpTYPE *outparam)
{

  // xP is x3 in the paper reference frame;
  // yP is R in the paper reference frame.

  // i1  is the first index before the particle location, along x
  // j1  is the first index before the particle location, along y

  fpTYPE Lx = xx[Nx-1] - xx[0]; 
  fpTYPE Ly = yy[Ny-1] - yy[0]; 

  size_t i = floor((xP - xx[0]) / Lx * Nx) + 1;
  size_t j = floor((yP - yy[0]) / Ly * Ny) + 1;

  // Make sure the index is in the range (maybe particle went crazy!)
  if(i < 1)   {
    i = 0;
  } else if(i >= Nx-1)   {
    i = Nx-2;
  } 

  if(j < 1)  {
    j = 0;
  } else if(j >= Ny-1)   {
    j = Ny-2;
  }

  // Find the vector-ID for nodes
  size_t id_a = (i+1)*Ny + (j+1);
  size_t id_b = (i+1)*Ny + j;
  size_t id_c = (i)*Ny + j;
  size_t id_d = (i)*Ny + (j+1);

  // Compute weights
  fpTYPE A = (xP - xx[i])*(yP - yy[j]);
  fpTYPE B = (xP - xx[i])*(yy[j+1] - yP);
  fpTYPE C = (xx[i+1] - xP)*(yy[j+1] - yP);
  fpTYPE D = (xx[i+1] - xP)*(yP - yy[j]);

  // Interpolation
  outparam[0] = (A*N_MAT[id_a]  + B*N_MAT[id_b]  + C*N_MAT[id_c]  + D*N_MAT[id_d])  / (A+B+C+D); // Number density
  outparam[1] = (A*V1_MAT[id_a] + B*V1_MAT[id_b] + C*V1_MAT[id_c] + D*V1_MAT[id_d]) / (A+B+C+D); // V1 (v_R)
  outparam[2] = (A*V2_MAT[id_a] + B*V2_MAT[id_b] + C*V2_MAT[id_c] + D*V2_MAT[id_d]) / (A+B+C+D); // V2 (v_theta)
  outparam[3] = (A*V3_MAT[id_a] + B*V3_MAT[id_b] + C*V3_MAT[id_c] + D*V3_MAT[id_d]) / (A+B+C+D); // V3 (v_z)
  outparam[4] = (A*T_MAT[id_a]  + B*T_MAT[id_b]  + C*T_MAT[id_c]  + D*T_MAT[id_d])  / (A+B+C+D); // Temperature

  return;
}

// ##############################################################
 
__device__ void collision_rotate_vel(fpTYPE& v1, fpTYPE& v2, fpTYPE& v3, fpTYPE* v_collpart, fpTYPE alpha, hiprandState *state)
{
  // Compute relative velocity
  fpTYPE g1 = v_collpart[0] - v1;
  fpTYPE g2 = v_collpart[1] - v2;
  fpTYPE g3 = v_collpart[2] - v3;

  fpTYPE g = sqrt(g1*g1 + g2*g2 + g3*g3);

  // Perform collision!
  fpTYPE q = 2.0*pow(rnum(state), 1.0/alpha) - 1.0;
  fpTYPE cos_th = q;
  fpTYPE sin_th = sqrt(1.0 - q*q);
  
  fpTYPE chi = 2*PI*rnum(state);
  fpTYPE cos_chi = cos(chi);
  fpTYPE sin_chi = sin(chi);
  
  fpTYPE g1_prime = g*sin_th*cos_chi;
  fpTYPE g2_prime = g*sin_th*sin_chi;
  fpTYPE g3_prime = g*cos_th;

  v1 = v_collpart[0] + g1_prime;
  v2 = v_collpart[1] + g2_prime;
  v3 = v_collpart[2] + g3_prime;

  return;
}

// ############################################################################

__global__ void myKernel(hiprandState *state, int pPERt, size_t Nx_BG, size_t Ny_BG, 
                         fpTYPE* d_xx, fpTYPE* d_yy, 
                         fpTYPE* d_N_BG, fpTYPE* d_V1_BG, fpTYPE* d_V2_BG, fpTYPE* d_V3_BG, fpTYPE* d_T_BG,
                         fpTYPE* d_times,
                         fpTYPE* d_x1P, fpTYPE* d_x2P, fpTYPE* d_x3P,
                         fpTYPE  z_slit_start)
{

  fpTYPE x1, x2, x3;
  fpTYPE v1, v2, v3;
  fpTYPE v1_prime, v2_prime;
  fpTYPE R, th;

  // Simulation parameters
  fpTYPE Ldomain  = 0.025;   // [m] length of domain (channel)
  fpTYPE R_chan_ext = 0.05;  // [m] radius of external wall
  fpTYPE R_chan_int = 0.035; // [m] radius of internal wall

  fpTYPE Twall  = 300.0; // [K]
  fpTYPE Tanode = 300.0; // [K]
  fpTYPE Tinj   = 300.0; // [K]
  fpTYPE M     = 2.18e-25;  // [kg] particles mass

  // // Uniform injection from anode 
  // fpTYPE u1_inj = 0.0; // uR
  // fpTYPE u2_inj = 0.0; // utheta
  // fpTYPE u3_inj = 0.0; // uz
 
  // Injection from slit 
  fpTYPE u1_inj = 88.975; // uR
  fpTYPE u2_inj = 0.0; // utheta
  fpTYPE u3_inj = -154.11; // uz

  int tID_glob = blockIdx.x*blockDim.x + threadIdx.x;

  for(size_t IDp = 0; IDp < pPERt; ++IDp)
  {
    // Unique identifier of particle
    size_t pID_global = IDp + tID_glob*pPERt;
    d_times[pID_global] = 0.0; // Init

    // // Inject particles from anode
    // x1 = rnum(state)*(R_chan_ext - R_chan_int)*0.98 + R_chan_int*1.001;
    // x2 = 0.0;
    // x3 = 0.0;

    // Inject particles from side walls
    x1 = R_chan_int;
    x2 = 0.0;
    x3 = rnum(state)*0.001 + z_slit_start; // 1 mm slit
  
    fpTYPE vel_now[3];
    maxwellian(u1_inj, u2_inj, u3_inj, Tinj, M, state, vel_now);
  
    v1 = vel_now[0];
    v2 = vel_now[1];
    v3 = vel_now[2];
  
    // Loop until particle stays inside the domain
    size_t counter = 0;
    while( x3 < Ldomain )
    {
      // std::cout << "Advecting..." << std::endl;
  
      // Timestep adjustment
  
      fpTYPE v_abs = sqrt(v1*v1 + v2*v2 + v3*v3); 
      fpTYPE dt = (Ldomain/(v_abs + 1.0e-5))/50.0; // so it takes 100 steps to do one full domain
  
      // Timestep adjustment
  
      // ++++++++ Advect particle ++++++++++++
      x1 += v1*dt;
      x2 += v2*dt;
      x3 += v3*dt;
  
      // Rotate back position into plane (cylindrical coordinates)
      R  = sqrt(x1*x1 + x2*x2);
      th = atan2(x2,x1); 
  
      x1 = R;
      x2 = 0;
  
      // Rotate velocity
      v1_prime = cos(th)*v1 + sin(th)*v2;
      v2_prime = -sin(th)*v1 + cos(th)*v2;
  
      v1 = v1_prime; // Updated velocity
      v2 = v2_prime;
  
      // +++++++++ Check boundaries ++++++++++
      if( R < R_chan_int ) // Inner wall hit
      {
        fpTYPE tOUT = abs( (x1 - R_chan_int)/(v1+1.0e-10) ); // How much time was spent out (spend it IN!)
  
        // Remove trajectory out of wall
        x1 -= tOUT*v1;
        x3 -= tOUT*v3;
  
        v1 = -1; // Init like this to enter the loop
        while (v1 < 0)
        {
          maxwellian(0.0, 0.0, 0.0, Twall, M, state, vel_now);
          v1 = vel_now[0];
          v2 = vel_now[1];
          v3 = vel_now[2];
        }
     
        // Add trajectory out of wall
        // x1 = R_chan_int + v1*tOUT; // Finish advection
        x1 += v1*tOUT;
        x3 += v3*tOUT;
  
      } 
      else if (R > R_chan_ext) // Outer wall hit
      {
        fpTYPE tOUT = abs( (x1 - R_chan_ext)/(v1+1.0e-10) ); // How much time was spent out (spend it IN!)
  
        // Remove trajectory out of wall
        x1 -= tOUT*v1;
        x3 -= tOUT*v3;
  
        v1 = +1; // Init like this to enter the loop
        while (v1 > 0)
        {
          maxwellian(0.0, 0.0, 0.0, Twall, M, state, vel_now);
          v1 = vel_now[0];
          v2 = vel_now[1];
          v3 = vel_now[2];
        }
      
        // Add trajectory out of wall
        // x1 = R_chan_ext + v1*tOUT; // Finish advection
        x1 += v1*tOUT;
        x3 += v3*tOUT;
  
      }
      else if (x3 < 0.0) // Back wall hit
      {
        fpTYPE tOUT = abs( x3/(v3+1.0e-10) ); // How much time was spent out (spend it IN!)
  
        // Remove trajectory out of wall
        x1 -= v1*tOUT;
        x3 -= v3*tOUT;
  
        v3 = -1; // Init like this to enter the loop
        while (v3 < 0)
        {
          maxwellian(0.0, 0.0, 0.0, Tanode, M, state, vel_now);
          v1 = vel_now[0];
          v2 = vel_now[1];
          v3 = vel_now[2];
        }
  
        // Add trajectory out of wall
        // x3 = v3*tOUT; // Finish advection
        x1 += v1*tOUT;
        x3 += v3*tOUT;
      }

      // ++++++++ Perform collisions +++++++++++

      // Interpolate parameters at local position
      fpTYPE params[5];
      interp2D(x3, R, d_xx, d_yy, d_N_BG, d_V1_BG, d_V2_BG, d_V3_BG, d_T_BG, Nx_BG, Ny_BG, params);
  
      fpTYPE n_bg_now  = params[0];
      fpTYPE u1_bg_now = params[1];
      fpTYPE u2_bg_now = params[2];
      fpTYPE u3_bg_now = params[3];
      fpTYPE T_bg_now  = params[4];

      // ***** Hard-Sphere cross section *****
      fpTYPE d_molec = 5.74e-10;
      fpTYPE sig = PI*d_molec*d_molec; // [m2] Cross section
      fpTYPE alpha = 1.0;  // Scattering parameter (Bird, 1994, Appendix A) = 1 for isotropic scattering
      fpTYPE vTH_rel = sqrt(8*KB*T_bg_now/PI/(M/2.0)); // relative thermal vel: use reduced mass!

      fpTYPE g1 = v1 - u1_bg_now; // Relative velocity
      fpTYPE g2 = v2 - u2_bg_now; // Relative velocity
      fpTYPE g3 = v3 - u3_bg_now; // Relative velocity

      fpTYPE nu = sig*n_bg_now/PI*sqrt(2*PI*KB*T_bg_now/M)*psi_HS(sqrt(g1*g1 + g2*g2 + g3*g3)*sqrt(M/(2*KB*T_bg_now)) );
      // SIMPLE CASE // fpTYPE nu = vTH_rel*n_bg_now*sig;// collision frequency
      // *****************************
      
      fpTYPE Pcoll = 1 - exp(-nu*dt);
  
      if ( rnum(state) < Pcoll ) // Collision happens?
      { 
        // Create colliding neutral from local features
        fpTYPE v_collpart[3];
        maxwellian(u1_bg_now, u2_bg_now, u3_bg_now, T_bg_now, M, state, v_collpart);

        // Perform collision
        collision_rotate_vel(v1, v2, v3, v_collpart, alpha, state); 
      }

      // DBDBDB - export some particles, for testing purposes
      d_x1P[counter] = x1;
      d_x2P[counter] = x2;
      d_x3P[counter] = x3;
      counter++;

      // Add timestep to particle residence time
      if( (x3 <= 0.018) && (x3 >= 0.013) ) {
        d_times[pID_global] += dt; // Add timestep
      }

    } // end while particle inside domain

  } // end loop on particles

}

// ############################################################################

int main()
{
  int NTH = 512; // number of threads per block
  // int NTH = 84; // number of threads per block
  int NB  = 60; // number of blocks
  int pPERt = 33; // particles per thread

  // // For testing
  // int NTH   = 1;
  // int NB    = 1;
  // int pPERt = 1;

  // std::cout << "We will simulate " << pPERt*NTH*NB << " particles." << std::endl;

  // ========= Setup PRNG on GPU ========================
  hiprandState *devStates;
  hipMalloc((void **)&devStates, NB*NTH*sizeof(hiprandState));

  // ========= Load background gas data =================
  fpTYPE z_slit_start = 0.015;
  
  // std::string dirname = "matrices_uniform/";
  std::string dirname = "matrices_rev30_15_COLD/";

  size_t Nx_BG, Ny_BG;
  std::ifstream f( (dirname + "MAT_N.dat").c_str() );  // Read first line of a random file
  f >> Nx_BG >> Ny_BG; // Read dimensions as first elements, (Nx  Ny)

  // Allocate host memory
  fpTYPE *h_xx = new fpTYPE[Nx_BG];
  fpTYPE *h_yy = new fpTYPE[Ny_BG];

  fpTYPE *h_N_BG  = new fpTYPE[Nx_BG*Ny_BG];
  fpTYPE *h_V1_BG = new fpTYPE[Nx_BG*Ny_BG];
  fpTYPE *h_V2_BG = new fpTYPE[Nx_BG*Ny_BG];
  fpTYPE *h_V3_BG = new fpTYPE[Nx_BG*Ny_BG];
  fpTYPE *h_T_BG  = new fpTYPE[Nx_BG*Ny_BG];

  // Load background gas data into host arrays
  FillVectorFile(h_xx,    (dirname + "xx.dat").c_str());
  FillVectorFile(h_yy,    (dirname + "yy.dat").c_str());
  FillMatrixFile(h_N_BG,  (dirname + "MAT_N.dat").c_str());
  FillMatrixFile(h_V1_BG, (dirname + "MAT_V.dat").c_str()); // "y" in the DSMC simulation
  FillMatrixFile(h_V2_BG, (dirname + "MAT_W.dat").c_str()); // "z" in the DSMC simulation
  FillMatrixFile(h_V3_BG, (dirname + "MAT_U.dat").c_str()); // "x" in the DSMC simulation
  FillMatrixFile(h_T_BG,  (dirname + "MAT_T.dat").c_str());

  // Init device variables
  fpTYPE *d_xx; 
  fpTYPE *d_yy; 

  fpTYPE *d_N_BG; 
  fpTYPE *d_V1_BG;
  fpTYPE *d_V2_BG;
  fpTYPE *d_V3_BG;
  fpTYPE *d_T_BG; 

  hipMalloc(&d_xx, sizeof(fpTYPE)*Nx_BG);
  hipMalloc(&d_yy, sizeof(fpTYPE)*Ny_BG);

  hipMalloc(&d_N_BG,  sizeof(fpTYPE)*Nx_BG*Ny_BG);
  hipMalloc(&d_V1_BG, sizeof(fpTYPE)*Nx_BG*Ny_BG);
  hipMalloc(&d_V2_BG, sizeof(fpTYPE)*Nx_BG*Ny_BG);
  hipMalloc(&d_V3_BG, sizeof(fpTYPE)*Nx_BG*Ny_BG);
  hipMalloc(&d_T_BG,  sizeof(fpTYPE)*Nx_BG*Ny_BG);

  // Copy stuff into GPU global memory
  hipMemcpy(d_xx, h_xx, sizeof(fpTYPE)*Nx_BG, hipMemcpyHostToDevice);
  hipMemcpy(d_yy, h_yy, sizeof(fpTYPE)*Ny_BG, hipMemcpyHostToDevice);

  hipMemcpy(d_N_BG,  h_N_BG,  sizeof(fpTYPE)*Nx_BG*Ny_BG, hipMemcpyHostToDevice);
  hipMemcpy(d_V1_BG, h_V1_BG, sizeof(fpTYPE)*Nx_BG*Ny_BG, hipMemcpyHostToDevice);
  hipMemcpy(d_V2_BG, h_V2_BG, sizeof(fpTYPE)*Nx_BG*Ny_BG, hipMemcpyHostToDevice);
  hipMemcpy(d_V3_BG, h_V3_BG, sizeof(fpTYPE)*Nx_BG*Ny_BG, hipMemcpyHostToDevice);
  hipMemcpy(d_T_BG,  h_T_BG,  sizeof(fpTYPE)*Nx_BG*Ny_BG, hipMemcpyHostToDevice);

  // ========= Call kernel =================
  fpTYPE *h_times = new fpTYPE[NB*NTH*pPERt]; // Total number of particles
  fpTYPE *d_times;
  hipMalloc(&d_times, sizeof(fpTYPE)*NB*NTH*pPERt);

  size_t Ntest = 100000;
  fpTYPE *h_x1P = new fpTYPE[Ntest];
  fpTYPE *h_x2P = new fpTYPE[Ntest];
  fpTYPE *h_x3P = new fpTYPE[Ntest];

  fpTYPE *d_x1P; 
  fpTYPE *d_x2P; 
  fpTYPE *d_x3P; 

  hipMalloc(&d_x1P, sizeof(fpTYPE)*Ntest);
  hipMalloc(&d_x2P, sizeof(fpTYPE)*Ntest);
  hipMalloc(&d_x3P, sizeof(fpTYPE)*Ntest);

  // for(size_t slitID = 0; slitID < 24; slitID++) {
  //   fpTYPE z_slit_start = 0.024  - slitID*0.001;

    initRNG_kernel<<<NB,NTH>>>(devStates);
    myKernel<<<NB,NTH>>>(devStates, pPERt, Nx_BG, Ny_BG, d_xx, d_yy, d_N_BG, d_V1_BG, d_V2_BG, d_V3_BG, d_T_BG, d_times, d_x1P, d_x2P, d_x3P, z_slit_start);
    
    hipMemcpy(h_times,  d_times,  sizeof(fpTYPE)*NB*NTH*pPERt, hipMemcpyDeviceToHost);
  
    // // Print residence times now
    // for(size_t IDp = 0; IDp < NB*NTH*pPERt; ++IDp)
    // {
    //   std::cout << h_times[IDp] << std::endl;
    // }
  
    // Compute average residence time
    fpTYPE tau_ave = 0.0;
    
    for(size_t IDp = 0; IDp < NB*NTH*pPERt; ++IDp)
    {
      tau_ave += h_times[IDp]/(NB*NTH*pPERt);
    }
    
    std::cout << z_slit_start << "  " << tau_ave << std::endl;

  // }

  // hipMemcpy(h_x1P,  d_x1P,  sizeof(fpTYPE)*Ntest, hipMemcpyDeviceToHost);
  // hipMemcpy(h_x2P,  d_x2P,  sizeof(fpTYPE)*Ntest, hipMemcpyDeviceToHost);
  // hipMemcpy(h_x3P,  d_x3P,  sizeof(fpTYPE)*Ntest, hipMemcpyDeviceToHost);

  // for (size_t iii = 0; iii < Ntest; ++iii)
  //   std::cout << h_x1P[iii] << " " << h_x2P[iii] << " " << h_x3P[iii] << std::endl;

  // ======== Freed memory =================
  hipFree(devStates);

  hipFree(d_xx);
  hipFree(d_yy);

  hipFree(d_N_BG); 
  hipFree(d_V1_BG);
  hipFree(d_V2_BG);
  hipFree(d_V3_BG);
  hipFree(d_T_BG); 
  
  delete[] h_xx;
  delete[] h_yy;

  delete[] h_N_BG; 
  delete[] h_V1_BG;
  delete[] h_V2_BG;
  delete[] h_V3_BG;
  delete[] h_T_BG; 

  hipFree(d_times);
  delete[] h_times;

  // TESTING 
  hipFree(d_x1P); 
  hipFree(d_x2P); 
  hipFree(d_x3P); 
  delete[] h_x1P;
  delete[] h_x2P;
  delete[] h_x3P;


  return 0;
}
